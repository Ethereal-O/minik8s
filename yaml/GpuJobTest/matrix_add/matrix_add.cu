#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void mat_add(int *dmat_a,int* dmat_b,int *dmat_c,int rowNum,int colNum){
    int idx_x=blockIdx.x*blockDim.x+threadIdx.x;
    int idx_y=blockIdx.y*blockDim.y+threadIdx.y;
	int idx=idx_x+idx_y*colNum;
	if(idx<rowNum*colNum){
	    dmat_c[idx]=dmat_a[idx]+dmat_b[idx];
	}
}

int create_rand(){
        int ret = rand() % 10 + 1;
	return ret;
}

void mat_init(int *mat,int rowNum,int colNum){
    for(int i=0;i<rowNum;i++){
    	for(int j=0;j<colNum;j++){
    		mat[i*colNum+j]=create_rand();
    	}
    }
}

void print_result(int *mat,int m,int n){
	for(int i=0;i<m;i++){
                for(int j=0;j<n;j++){
                        printf("%d\t",mat[i*n+j]);
                }
		printf("\n");
        }
}

int main(){
	int m=1<<3,n=1<<4;
	int thread_per_block_x=1<<3;
	int thread_per_block_y=1<<3;
	dim3 block((n+thread_per_block_x-1)/thread_per_block_x,
	            (m+thread_per_block_y-1)/thread_per_block_y);
    dim3 thread(thread_per_block_x,thread_per_block_y);
	int *mat_a,*mat_b,*mat_c;
	int *dmat_a,*dmat_b,*dmat_c;

	mat_a=(int*)calloc(m*n,sizeof(int));
	mat_b=(int*)calloc(m*n,sizeof(int));
	mat_c=(int*)calloc(m*n,sizeof(int));

	hipMalloc((void**)&dmat_a,m*n*sizeof(int));
	hipMalloc((void**)&dmat_b,m*n*sizeof(int));
	hipMalloc((void**)&dmat_c,m*n*sizeof(int));

	mat_init(mat_a,m,n);
	mat_init(mat_b,m,n);

	hipMemcpy(dmat_a,mat_a,m*n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dmat_b,mat_b,m*n*sizeof(int),hipMemcpyHostToDevice);

    mat_add<<<block,thread>>>(dmat_a,dmat_b,dmat_c,m,n);

    hipMemcpy(mat_c,dmat_c,m*n*sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

    printf("mat_a:\n");
	print_result(mat_a,m,n);
    printf("mat_b:\n");
	print_result(mat_b,m,n);
    printf("mat_c:\n");
	print_result(mat_c,m,n);

	free(mat_a);
	free(mat_b);
	free(mat_c);

	hipFree(dmat_a);
	hipFree(dmat_b);
	hipFree(dmat_c);
}
