#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void mat_mul(int *dmat_a,int* dmat_b,int *dmat_c,int rowNum,int colNum,int midNum){
    int idx_x=blockIdx.x*blockDim.x+threadIdx.x;
    int idx_y=blockIdx.y*blockDim.y+threadIdx.y;
	int idx=idx_x+idx_y*colNum;
	if(idx<rowNum*colNum){
	    int res=0;
	    for(int i=0;i<midNum;i++){
	        res+=dmat_a[idx_y*midNum+i]*dmat_b[i*colNum+idx_x];
	    }
	    dmat_c[idx]=res;
	}
}

int create_rand(){
        int ret = rand() % 10 + 1;
	return ret;
}

void mat_init(int *mat,int rowNum,int colNum){
    for(int i=0;i<rowNum;i++){
    	for(int j=0;j<colNum;j++){
    		mat[i*colNum+j]=create_rand();
    	}
    }
}

void print_result(int *mat,int rowNum,int colNum){
	for(int i=0;i<rowNum;i++){
                for(int j=0;j<colNum;j++){
                        printf("%d\t",mat[i*colNum+j]);
                }
		printf("\n");
        }
}

int main(){
	int m=1<<5,n=1<<6,p=1<<3;
	int thread_per_block_x=1<<3;
	int thread_per_block_y=1<<3;
	dim3 block((p+thread_per_block_x-1)/thread_per_block_x,
	            (m+thread_per_block_y-1)/thread_per_block_y);
    dim3 thread(thread_per_block_x,thread_per_block_y);
	int *mat_a,*mat_b,*mat_c;
	int *dmat_a,*dmat_b,*dmat_c;

	mat_a=(int*)calloc(m*n,sizeof(int));
	mat_b=(int*)calloc(n*p,sizeof(int));
	mat_c=(int*)calloc(m*p,sizeof(int));

	hipMalloc((void**)&dmat_a,m*n*sizeof(int));
	hipMalloc((void**)&dmat_b,n*p*sizeof(int));
	hipMalloc((void**)&dmat_c,m*p*sizeof(int));

	mat_init(mat_a,m,n);
	mat_init(mat_b,n,p);

	hipMemcpy(dmat_a,mat_a,m*n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dmat_b,mat_b,n*p*sizeof(int),hipMemcpyHostToDevice);

    mat_mul<<<block,thread>>>(dmat_a,dmat_b,dmat_c,m,p,n);

    hipMemcpy(mat_c,dmat_c,m*p*sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

    printf("mat_a:\n");
	print_result(mat_a,m,n);
    printf("mat_b:\n");
	print_result(mat_b,n,p);
    printf("mat_c:\n");
	print_result(mat_c,m,p);

	free(mat_a);
	free(mat_b);
	free(mat_c);

	hipFree(dmat_a);
	hipFree(dmat_b);
	hipFree(dmat_c);
}
