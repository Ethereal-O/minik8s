#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void mat_mul(int *dmat_a,int* dmat_b,int *dmat_c,int rowNum,int colNum,int midNum){
    int idx_x=blockIdx.x*blockDim.x+threadIdx.x;
    int idx_y=blockIdx.y*blockDim.y+threadIdx.y;
	int idx=idx_x+idx_y*colNum;
	if(idx<rowNum*colNum){
	    int res=0;
	    for(int i=0;i<midNum;i++){
	        res+=dmat_a[idx_y*midNum+i]*dmat_b[i*colNum+idx_x];
	    }
	    dmat_c[idx]=res;
	}
}

int create_rand(){
        int ret = rand() % 10 + 1;
	return ret;
}

void mat_init(int *mat,int rowNum,int colNum){
    for(int i=0;i<rowNum;i++){
    	for(int j=0;j<colNum;j++){
    		mat[i*colNum+j]=create_rand();
    	}
    }
}

void judge_result(int *mat_cpu,int *mat_gpu,int rowNum,int colNum){
	for(int i=0;i<rowNum;i++){
                for(int j=0;j<colNum;j++){
                        if(mat_cpu[i*colNum+j]!=mat_gpu[i*colNum+j]){
                            printf("result error!\n");
                            return;
                        }
                }
        }
    printf("result pass!\n");
}



int main(){
	int m=1<<8,n=1<<8,p=1<<8;
	int thread_per_block_x=1<<4;
	int thread_per_block_y=1<<4;
	dim3 block((p+thread_per_block_x-1)/thread_per_block_x,
	            (m+thread_per_block_y-1)/thread_per_block_y);
    dim3 thread(thread_per_block_x,thread_per_block_y);
	int *mat_a,*mat_b,*mat_gpu,*mat_cpu;
	int *dmat_a,*dmat_b,*dmat_c;
	clock_t cpu_start,cpu_end,gpu_start,gpu_end;

	mat_a=(int*)calloc(m*n,sizeof(int));
	mat_b=(int*)calloc(n*p,sizeof(int));
	mat_gpu=(int*)calloc(m*p,sizeof(int));
	mat_cpu=(int*)calloc(m*p,sizeof(int));

	hipMalloc((void**)&dmat_a,m*n*sizeof(int));
	hipMalloc((void**)&dmat_b,n*p*sizeof(int));
	hipMalloc((void**)&dmat_c,m*p*sizeof(int));

	mat_init(mat_a,m,n);
	mat_init(mat_b,n,p);

    cpu_start=clock();
    for(int i=0;i<m;i++){
        for(int j=0;j<p;j++){
            int res=0;
            for(int k=0;k<n;k++){
               res+=mat_a[i*n+k]*mat_b[k*p+j];
            }
            mat_cpu[i*p+j]=res;
        }
    }
    cpu_end=clock();

	hipMemcpy(dmat_a,mat_a,m*n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dmat_b,mat_b,n*p*sizeof(int),hipMemcpyHostToDevice);

	gpu_start=clock();
    mat_mul<<<block,thread>>>(dmat_a,dmat_b,dmat_c,m,p,n);

    hipMemcpy(mat_gpu,dmat_c,m*p*sizeof(int),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gpu_end=clock();

	printf("cpu : %d %d<-> gpu: %d %d\n",cpu_end,cpu_start,gpu_end,gpu_start);
	judge_result(mat_cpu,mat_gpu,m,p);


	free(mat_a);
	free(mat_b);
	free(mat_gpu);
	free(mat_cpu);

	hipFree(dmat_a);
	hipFree(dmat_b);
	hipFree(dmat_c);
}