#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void add(int *a) {
        a[0]=a[0]*2;
}

int main() {
    int N = 1;
    int *a;
    int *dev_a; 

    a = (int*)malloc(N * sizeof(int)); 
    hipMalloc((void**)&dev_a, N * sizeof(int));   
    
    a[0]=100;
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    printf("a[0] = %d\n", a[0]);
    add<<<1, 1>>>(dev_a);
    hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("a[0] = %d\n", a[0]);

    free(a);
    hipFree(dev_a);
    return 0;
}

